#pragma once

#include "runConvection2Dparallel.cuh"

#define N 32

void runConvection2Dparallel()
{
	// Declaring on host
	double A[N][N];
	double B[N][N];
	double C[N][N];

	double d = double(2) / N;
	int nt = 10;
	double c = 0.2;
	double sigma = 0.2;
	double dt = sigma * d;

	hipError_t cudaStatus;

	// Device pointers
	double(*d_A)[N], (*d_B)[N], (*d_C)[N];

	// Allocate host memory
	cudaStatus = hipMalloc((void**)&d_A, N * N * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&d_B, N * N * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&d_C, N * N * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	// Copy to device from host
	cudaStatus = hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	cudaStatus = hipMemcpy(d_B, B, N * N * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	cudaStatus = hipMemcpy(d_C, C, N * N * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	// One block only
	int blocks = 1;

	// One thread per element
	dim3 threads(N, N);

	stepFunction2D << <blocks, threads >> > (d_A, d);

	for (int t = 0; t < nt; t++) {
		matrixCopy << <blocks, threads >> > (d_A, d_B);
		finiteDiff2D << <blocks, threads >> > (d_A, d_B, d, dt, c);
	}

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
	}

	// Copy back to host from device
	cudaStatus = hipMemcpy(C, d_C, N * N * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	cudaStatus = hipMemcpy(A, d_A, N * N * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}